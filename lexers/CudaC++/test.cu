
#include <hip/hip_runtime.h>
#include <cstdio>

extern "C" {
  __global__ void helloWorld(char *data) {
    #if __CUDA_ARCH__ >= 200
    printf("Hello, world! I'm thread (%d,%d,%d) in block (%d,%d,%d).\n",
      threadIdx.x, threadIdx.y, threadIdx.z,
      blockIdx.x, blockIdx.y, blockIdx.z);
    #endif

    int sum = 0;
    for (int i=0; i<100; i++) {
      sum += data[i];
    }
    
    #if __CUDA_ARCH__ >= 200
    printf("The sum is: %d\n", sum);
    #endif
  }
}
