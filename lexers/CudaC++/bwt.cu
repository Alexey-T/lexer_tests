#ifndef BWT_CU
#define BWT_CU
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/copy.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/fill.h>

#include <cstdio>
#include <iostream>
#include <cstring>
#include <iterator>

#include "../lib/device_string.cu"

#define POOL_SZ (10*1024*1024)

using namespace std;

void rotate(int N, char *word, vector<string> *h_vec)
{
	char *str, *rot;
	hipMalloc((void**)&str, /*sizeof(char) * */(N + 1));
	hipMalloc((void**)&rot, /*sizeof(char) * */((N + 1) * (N + 1)));
		
	thrust::device_ptr<char> strD(str);
	thrust::device_ptr<char> rotD(rot);
	thrust::copy(word, word + N, strD);
	
	for (int i = 0; i < N; i++)	//Rotations happen in this loop
	{
		thrust::copy(strD + i, strD + N, rotD + (i * N));
		thrust::copy(strD, strD + i, rotD + (i * N) + (N - i));
	}
	
	for (int i = 0; i < N; i++)	//We extract data back from the GPU
	{
		hipMemcpy(word, rot + (i * N), N, hipMemcpyDeviceToHost);
		h_vec->push_back(word);
	}
	
	hipFree(str);
	hipFree(rot);
}

void sort(vector<string> *h_vec, char *result)
{
	thrust::device_vector<device_string> d_vec;
	d_vec.reserve(h_vec->size());

	for(vector<std::string>::iterator iter = h_vec->begin(); iter!=h_vec->end(); ++iter)
	{
		device_string d_str(*iter);
		d_vec.push_back(d_str);
	}

	thrust::sort(d_vec.begin(), d_vec.end() );
	
	for(int i = 0; i < d_vec.size(); i++)
	{
		device_string d_str(d_vec[i]);
		h_vec->at(i) = d_str;
		result[i] = (h_vec->at(i)).at(h_vec->at(i).length() - 1);
	}
}

char* bwt( char *word)
{
	int N = strlen(word);
	vector<string> h_vec;

	rotate(N, word, &h_vec);

	char *result = new char(N);	
	sort(&h_vec, result);	
	
	return result;
}

int main(int argc, char *argv[])
{	
	if (argc != 2)
	{
		cout << "Usage: bwt.out STRING_INPUT" << endl;
		exit(1);
	}

//	char word[256];
	int N = strlen(argv[1]);
//	char * word = new char(N);
//	strncpy(word, bwt(argv[1]), N);	
//	cout << word << endl;
	cout << bwt(argv[1]) << endl;	
	return 0;
}

#endif
