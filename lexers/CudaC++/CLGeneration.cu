#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/copy.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/fill.h>

#include <cstdio>
#include <iostream>
#include <cstring>
#include <iterator>

void clgeneration()
{
	//Initialisation
	thrust::device_vector<char> d_list(256);
	thrust::sequence(d_list.begin(), d_list.begin() + 256);
	thrust::device_vector<int> d_freq(256);
//	thrust::sort_by_key(d_freq.begin(), d_freq.end(), d_list);
	thrust::device_vector<int> d_leader(256);
	thrust::device_vector<int> cl(256);
	thrust::fill(d_leader.begin(), d_leader.end(), -1);
	thrust::fill(cl.begin(), cl.end(), 0);
	thrust::device_vector<int> front = cl.begin();
	thrust::device_vector<int> rear = cl.begin();
	thrust::device_vector<int> curr = cl.begin();

	//New iNode
	thrust::device_vector<int> mid(4);
	thrust::device_vector<int> MinFreq;
	thrust::fill(mid.begin(), mid.end(), 500);
	if (curr ≤ 255)
		mid [0] = d_freq[curr+1];
/*	if (curr ≤ 254)
		mid [1] = d_freq[curr+2];
	if (rear > front)
		mid [2] = d_freq[front+1];
	if (rear > front + 1)
		mid [3] = d_freq[front+2];
	MinFreq = mid[0] + mid[1];
	d_freq[rear + 1] = MinFreq;
	d_leader[rear + 1] = -1;
	if (isLeaf (mid[0]))
		leader[curr+1] = rear + 1;
		cl[curr+1] = cl[curr + 1] + 1;
		curr = curr + 1;
	else
		leader[front + 1] = rear + 1;
		front = front + 1;
	if (isLeaf(mid[1]))
		leader[curr + 1] = rear + 1;
		cl[curr+1] = cl[curr + 1] + 1;
		curr = curr + 1;
	else
		leader[front + 1] = rear + 1;

//Select Module
Forall processors Pi (lNodesCur < i ≤ n)
	if (d_freq[i] ≤ MinFreq)
		Copy[i – lNodesCur].freq ← lNodes[i].freq
		Copy[i – lNodesCur].index ← i
		Copy[i – lNodesCur].isLeaf ← true
		if (i = n || lNodes[i+1].freq > MinFreq)
			CurLeavesNum ← i – lNodesCur

//Updating iterators
P1 Sets
	mergeRear ← iNodesRear
	mergeFront ← iNodesFront
	if((CurLeavesNum+ iNodesRear - iNodesFront)%2=0)
		iNodesFront ← iNodesRear
	else if ((iNodesRear - iNodesFront != 0) &&
	(F[lNodesCur+CurLeavesNum]≤iNodes[iNodesRear].freq))
		mergeRear--
		iNodesFront ← iNodesRear - 1
	else
		iNodesFront ← iNodesRear
		CurLeavesNum --
	lNodesCur ← lNodesCur + CurLeavesNum iNodesRear++

//Meld Module
Forall processors Pi (1 ≤ i ≤ TempLength) do in parallel
	ind ← iNodesRear + i
	iNodes [ind].freq ← temp [2*i-1].freq + temp [2*i].freq
	iNodes[ind].leader ← -1
	if (temp [2*i-1].isleaf)
		lNodes [temp [2*i – 1].index].leader ← ind
		CL[temp [2*i – 1].index]++
	else
		iNodes [temp [2*i – 1].index].leader ← ind
	if (temp [2*i].isleaf)
		lNodes [temp [2*i].index].leader ← ind
		CL[temp [2*i ].index]++
	else
		iNodes [temp [2*i].index].leader ← ind
P1 sets
	iNodesRear ← iNodesRear + (TempLength/2)


//Updating leaders
Forall processors Pi (1 ≤ i ≤ n) do in parallel
	if (lNodes[i].leader != -1)
		if (iNodes[lNodes[i].leader].leader != -1)
			lNodes[i].leader ← iNodes[lNodes[i].leader].leader
			CL[i] ++
*/
}

int main()
{
	clgeneration();
	return 0;
}
