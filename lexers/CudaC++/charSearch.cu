
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

__global__ void fnSearch(char *str, char *key, int *res)
{
	*res = -1;
	if(str[threadIdx.x] == *key)
		*res = threadIdx.x;
}

int main(int argc, char *argv[])
{
	if (argc != 3)
	{
		cout << "Usage: charSearch.out STRING KEY" << endl;
		exit(1);
	}

	char *dStr, *dKey;
	int *dRes, *hRes;
	hipMalloc((void**)&dStr, sizeof(char) * strlen(argv[1]));
	hipMalloc((void**)&dKey, sizeof(char));
	hipMalloc((void**)&dRes, sizeof(int));
	hRes = new(int);
	
	hipMemcpy(dStr, argv[1], sizeof(char) * strlen(argv[1]), hipMemcpyHostToDevice);
	hipMemcpy(dKey, argv[2], sizeof(char), hipMemcpyHostToDevice);
	
	fnSearch<<<1, strlen(argv[1])>>>(dStr, dKey, dRes);
	
	hipMemcpy(hRes, dRes, sizeof(int), hipMemcpyDeviceToHost);
	
	cout << "Result: " << *hRes << endl;
	
	return 0;
}
