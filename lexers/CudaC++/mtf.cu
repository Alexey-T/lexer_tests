#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/copy.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/find.h>

#include <cstdio>
#include <iostream>
#include <cstring>
#include <vector>

using namespace std;

__global__ void fnSearch(char *str, char *key, int *res)
{
	*res = -1;
	if(str[threadIdx.x] == *key)
		*res = threadIdx.x;
}

void mtf(vector<char> word)
{
	//Parallel initialisation of character set	
	thrust::device_vector<char> d_list(256);
	thrust::sequence(d_list.begin(), d_list.begin() + 256);
	thrust::host_vector<char> list(256);
	thrust::device_vector<char> d_word(word.size());
	thrust::device_vector<int> dRes;
	int counter;
	thrust::device_vector<char>::iterator iter, count;
	thrust::host_vector<char> h_word(word.size());
	char ch;
	h_word = word;
	d_word = h_word;

	int i;

	for (counter = 0; counter < word.size(); counter++)
	{
		thrust::device_vector<char> d_temp_b(255);
		thrust::copy(list.begin(), list.end(), d_list.begin());

		//Scan for character on cpu
		h_word[0] = d_word[counter];
		iter = thrust::find(d_list.begin(), d_list.end(), d_word[counter]);

		//Shifting of the character set in parallel
		if (d_list[0] != h_word[0])
		{
			thrust::copy(d_list.begin(), iter, list.begin()+1);
			list[0] = h_word[0];
		}
	}

	thrust::copy(list.begin(), list.end(), d_list.begin());
	thrust::copy(word.begin(), word.end(), d_word.begin());
	for (counter = 0; counter < list.size(); counter++)
	{
		iter = thrust::find(d_word.begin(), d_word.end(), d_list[counter]);
		while (iter != d_word.end())
		{
			*iter = counter;
			iter = thrust::find(d_word.begin(), d_word.end(), d_list[counter]);
		}
	}
	thrust::copy(d_word.begin(), d_word.end(), h_word.begin());

	for (counter = 0; counter < word.size(); counter++)
	{
		ch = h_word[counter];		
		cout << counter << "\t" << ch << endl;
	}
}

int main(int argc, char *argv[])
{
	if (argc != 2)
	{
		cout << "Usage: mtf.out STRING_INPUT" << endl;
		exit(1);
	}

	int len = strlen(argv[1]);
	vector<char> word(argv[1], argv[1] + len);
//	time_t begin, end;
//	begin = time(NULL);
//	for (int i = 0; i < 10000; i++)
		mtf(word);
//	end = time(NULL);
//	cout <<difftime(end, begin);
	return 0;
}
