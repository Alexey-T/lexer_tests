#ifndef STRING_SORT_CU
#define STRING_SORT_CU

//Taken from https://groups.google.com/group/thrust-users/msg/0eac80d2e41cbcfb?pli=1, https://groups.google.com/group/thrust-users/browse_thread/thread/f4b1b825cc927df9?pli=1, http://ldn.linuxfoundation.org/article/c-gpu-and-thrust-strings-gpu

//Our thanks to Shashank Srikant

#include <cstring>
#include <string>
#include <vector>
#include <iterator>

#include <thrust/device_ptr.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/copy.h>

#define POOL_SZ (10*1024*1024)

using namespace std;

extern "C" class device_string
{
public:
	int cstr_len;
	char* raw;
	thrust::device_ptr<char> cstr;

	static char* pool_raw;
	static thrust::device_ptr<char> pool_cstr;
	static thrust::device_ptr<char> pool_top;

	// Sets the variables up the first time its used.
	__host__ static void init()
	{
		static bool v = true;
		if( v )
		{
			v = false;

			pool_cstr = thrust::device_malloc(POOL_SZ);
			pool_raw  = (char*)raw_pointer_cast( pool_cstr );
			pool_top = pool_cstr;
		}
	}
	
	// Destructor for device variables used.
	__host__ static void fin()
	{
		init();
		thrust::device_free(pool_cstr);
	}

	// Parametrized constructor to copy one device_string to another.
	__host__ device_string( const device_string& s )
	{
		cstr_len = s.cstr_len;
		raw = s.raw;
		cstr = s.cstr;
	}

	// Parametrized constructor to copy a std::string to device_string type
	__host__ device_string( const std::string& s )
	{
		cstr_len = s.length();
		init();
		cstr = pool_top;
		pool_top += cstr_len+1;
		raw = (char *) raw_pointer_cast(cstr);
		hipMemcpy( raw, s.c_str(), cstr_len+1, hipMemcpyHostToDevice );
	}
	
	// Default constructor.
	__host__ __device__ device_string()
	{
		cstr_len = -1;
		raw = NULL;
	}

	// Conversion operator to copy device_string type to std::string
	// This is where the problem is
	__host__ operator std::string(void)
	{
		std::string ret;
		//device_ptr<char*>::iterator it = cstr.begin();
		thrust::copy(cstr, cstr+cstr_len, back_inserter(ret));
		return ret;
	}
};

char* device_string::pool_raw;
thrust::device_ptr<char> device_string::pool_cstr;
thrust::device_ptr<char> device_string::pool_top;

// User-defined comparison operator
bool __device__ operator< (device_string lhs, device_string rhs)
{
	char *l = lhs.raw;
	char *r = rhs.raw;

	for( ; *l && *r && *l==*r; )
	{
	++l;
	++r;
	}
	return *l < *r;
}
#endif
